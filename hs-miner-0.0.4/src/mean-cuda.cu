#if EDGEBITS >= 28
#include "tromp/mean_miner.cu"
#endif

#include <assert.h>
#include <unistd.h>
#include <stdbool.h>
#include <stdint.h>
#include "common.h"

int32_t
hs_mean_cuda_run(
  hs_options_t *options,
  uint8_t *solution,
  uint32_t *result,
  bool *match
) {
#if EDGEBITS >= 28
  uint8_t header[MAX_HEADER_SIZE];
  size_t header_len = options->header_len;
  uint32_t nonce = options->nonce;
  uint32_t range = 1;
  uint32_t device = options->device;
  uint8_t hash[32];
  uint8_t chash[32];

  memset(hash, 0xff, 32);

  if (header_len < MIN_HEADER_SIZE || header_len > MAX_HEADER_SIZE)
    return HS_EBADARGS;

  memcpy(header, options->header, header_len);

  int32_t device_count = 0;
  hipGetDeviceCount(&device_count);

  if (device_count < 0 || device >= device_count)
    return HS_ENODEVICE;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  meancu_trimparams params;

  if (options->range)
    range = options->range;

  if (options->trims)
    params.ntrims = ((int32_t)options->trims) & -2;

  if (options->threads) {
    params.genUtpb *= options->threads;
    params.genV.stage1tpb *= options->threads;
    params.genV.stage2tpb *= options->threads;
    params.trim.stage1tpb *= options->threads;
    params.trim.stage2tpb *= options->threads;
    params.rename[0].stage1tpb *= options->threads;
    params.rename[0].stage2tpb *= options->threads;
    params.rename[1].stage1tpb *= options->threads;
    params.rename[1].stage2tpb *= options->threads;
    params.trim3tpb *= options->threads;
    params.rename3tpb *= options->threads;
    params.genUtpb /= 100;
    params.genV.stage1tpb /= 100;
    params.genV.stage2tpb /= 100;
    params.trim.stage1tpb /= 100;
    params.trim.stage2tpb /= 100;
    params.rename[0].stage1tpb /= 100;
    params.rename[0].stage2tpb /= 100;
    params.rename[1].stage1tpb /= 100;
    params.rename[1].stage2tpb /= 100;
    params.trim3tpb /= 100;
    params.rename3tpb /= 100;
  }

  if (prop.maxThreadsPerBlock <= params.genUtpb
      || prop.maxThreadsPerBlock <= params.genV.stage1tpb
      || prop.maxThreadsPerBlock <= params.genV.stage2tpb
      || prop.maxThreadsPerBlock <= params.trim.stage1tpb
      || prop.maxThreadsPerBlock <= params.trim.stage2tpb
      || prop.maxThreadsPerBlock <= params.rename[0].stage1tpb
      || prop.maxThreadsPerBlock <= params.rename[0].stage2tpb
      || prop.maxThreadsPerBlock <= params.rename[1].stage1tpb
      || prop.maxThreadsPerBlock <= params.rename[1].stage2tpb
      || prop.maxThreadsPerBlock <= params.trim3tpb
      || prop.maxThreadsPerBlock <= params.rename3tpb) {
    return HS_EBADPROPS;
  }

  hipSetDevice(device);

  meancu_solver_ctx ctx(params);

  bool has_sol = false;

  *result = 0;
  *match = false;

  for (uint32_t r = 0; r < range; r++) {
    if (!options->running)
      break;

    ctx.setheadernonce((char *)header, header_len, nonce + r);

    uint32_t nsols = ctx.solve();

    for (uint32_t s = 0; s < nsols; s++) {
      uint32_t *sol = &ctx.sols[s * PROOFSIZE];
      int32_t rc = verify(sol, &ctx.trimmer->sip_keys);

      if (rc == POW_OK) {
        hs_hash_solution(sol, chash);

        if (memcmp(chash, hash, 32) <= 0) {
          *result = nonce + r;
          for (int32_t i = 0; i < PROOFSIZE; i++)
            hs_write_u32(&solution[i * 4], sol[i]);
          memcpy(hash, chash, 32);
          has_sol = true;
        }

        if (memcmp(chash, options->target, 32) <= 0) {
          *match = true;
          return HS_SUCCESS;
        }
      }
    }
  }

  if (has_sol)
    return HS_SUCCESS;

  return HS_ENOSOLUTION;
#else
  return HS_ENOSUPPORT;
#endif
}
