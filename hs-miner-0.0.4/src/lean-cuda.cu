#if EDGEBITS >= 5
#include "tromp/lean_miner.cu"
#endif
#include <unistd.h>
#include <stdbool.h>
#include "common.h"

int32_t
hs_lean_cuda_run(
  hs_options_t *options,
  uint8_t *solution,
  uint32_t *result,
  bool *match
) {
#if EDGEBITS >= 5
  uint8_t header[MAX_HEADER_SIZE];
  size_t header_len = options->header_len;
  uint32_t nonce = options->nonce;
  uint32_t range = 1;
  uint32_t device = options->device;
  uint32_t nthreads = 16384;
  uint32_t ntrims = 32;
  uint32_t tpb = 0;

  if (header_len < MIN_HEADER_SIZE || header_len > MAX_HEADER_SIZE)
    return HS_EBADARGS;

  memcpy(header, options->header, header_len);

  if (options->range)
    range = options->range;

  if (options->threads)
    nthreads = options->threads;

  if (options->trims)
    ntrims = options->trims;

  int32_t device_count;
  hipGetDeviceCount(&device_count);

  if (device_count < 0 || device >= device_count)
    return HS_ENODEVICE;

  hipSetDevice(device);

  *result = 0;
  *match = false;

  return lean_run(
    &options->running,
    nthreads,
    ntrims,
    tpb,
    nonce,
    range,
    header,
    header_len,
    options->target,
    solution,
    result,
    match
  );
#else
  return HS_ENOSUPPORT;
#endif
}
